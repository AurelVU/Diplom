#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "mainSolver.h"
/*
ToDo:
�������� �������� ������ � ���, ��� � ��� �������� �������������� ���������� �� i=0 �� �������������, ���������� ��� j. � ������� ���������� ����������� �� ������ �����������. ���� ���-�� ��������� � ����
*/
hipError_t solver(float dx, float Tb, float Tb0, int imax, int jmax, float dh, float** T, int k, int FaceArea, float delt);


__global__ void solverKernel(float dx, float Tb, float Tb0, int imax, int jmax, float dh, float* T, float* T2, int k, int FaceArea, float delt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= 0 && i < imax && j >= 0 && j < jmax)
    {
        float Tc, Te, Tw, Tn, Ts;
        float FluxC, FluxE, FluxW, FluxN, FluxS;

        Tc = T[i * jmax + j];
        dx = dh;
        
        if (i == imax - 1) { Te = Tb0; dx = dx / 2; }
        else
            Te = T[i * jmax + j];
        FluxE = (-k * FaceArea) / dx;

        if (i == 0) { Tw = Tb0; dx = dx / 2; }
        else
            Tw = T[i * jmax + j];
        FluxW = (-k * FaceArea) / dx;

        if (j == jmax - 1) { Tn = Tb0; dx = dx / 2; }
        else
            Tn = T[i * jmax + j + 1];
        FluxN = (-k * FaceArea) / dx;

        if (j == 0) { Ts = Tb; dx = dx / 2; }
        else
            Ts = T[i * jmax + j - 1];
        FluxS = (-k * FaceArea) / dx;

        FluxC = FluxE + FluxW + FluxN + FluxS;

        T2[i * jmax + j] = Tc + delt * (FluxC * Tc - (FluxE * Te + FluxW * Tw + FluxN * Tn + FluxS * Ts));
    }
}

int main()
{
    mainSolver* m = new mainSolver();
    m->create("dfsfdsdf");
    m->RunPhysic();
    

    float dx = 0;
    float Tb = 240;
    float Tb0 = 0;
    float delt = 0.2;
    int FaceArea = 1;
    int k = 1;
    int dh = 1;
    int imax = 40;
    int jmax = 40;
    float** T = new float* [imax];
    for (int i = 0; i < imax; i++)
    {
        T[i] = new float[jmax];
        for (int j = 0; j < jmax; j++)
        {
            //if (j == 0)
            T[i][j] = 0;
        }
    }

    for (int i = 0; i < 49; i++)
        m->RunPhysic();


    for (int i = 0; i < imax; i++)
    {
        for (int j = 0; j < jmax; j++)
            printf("%.2f ", m->T[i][j]);
        printf("\n");
    }
    printf("###############################################################################\n");
    

    hipError_t cudaStatus = solver(dx, Tb, Tb0, imax, jmax, dh, T, k, FaceArea, delt);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}



hipError_t solver(float dx, float Tb, float Tb0, int imax, int jmax, float dh, float** T, int k, int FaceArea, float delt)
{
    float* dev_T = 0;
    float* dev_T2 = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_T, imax * jmax * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_T2, imax * jmax * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    dim3 gridSize = dim3((imax + 31) / 32, (jmax + 31) / 32, 1);    //������ ������������� �����
    dim3 blockSize = dim3(32, 32, 1); //������ ������������� �����


    // Launch a kernel on the GPU with one thread for each element.
    for (int i = 0; i < 50; i++)
    {
        solverKernel << <gridSize, blockSize >> > (dx, Tb, Tb0, imax, jmax, dh, dev_T, dev_T2, k, FaceArea, delt);
        float* a = dev_T;
        dev_T = dev_T2;
        dev_T2 = a;
    }
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    for (int i = 0; i < imax; i++)
    {
        cudaStatus = hipMemcpy(T[i], dev_T + i * jmax, jmax * sizeof(float), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
    }
    for (int i = 0; i < imax; i++)
    {
        for (int j = 0; j < jmax; j++)
            printf("%.2f ", T[i][j]);
        printf("\n");
    }
    
Error:
    hipFree(dev_T);
    hipFree(dev_T2);

    return cudaStatus;
}